﻿//alperen

#include <chrono>
#include "hip/hip_runtime.h"
#include ""
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc.hpp>
#include <iostream>
#include <stdio.h>


extern void median_filter_wrapper(const cv::Mat& input, cv::Mat& output, bool shared);


int insertionSort(int* window, int kernel_size)
{
    int temp, i, j;
    for (i = 0; i < (kernel_size * kernel_size); i++) {
        temp = window[i];
        for (j = i - 1; j >= 0 && temp < window[j]; j--) {
            window[j + 1] = window[j];
        }
        window[j + 1] = temp;
    }
    return window[kernel_size * kernel_size / 2 + 1];
}



int sort_short(int* ptr, int kernel_size) {
    int temp[256] = { 0 };

    for (int i = 0; i < kernel_size * kernel_size; i++) {
        temp[ptr[i]]++;
    }
    int counter = 0;


    for (int i = 0; i < 256; i++) {
        counter += temp[i];

        if (counter >= kernel_size * kernel_size /2+1) {
            return i;
        }
    }
    return 0;
}



cv::Mat cpu_edition(cv::Mat img, int sort_type, int kernel_size){
    cv::Mat final_img;
    int* window = new int(kernel_size * kernel_size);

    final_img = img.clone();

    for (int y = 0; y < img.rows; y++)
        for (int x = 0; x < img.cols; x++)
            final_img.at<uchar>(y, x) = 0.0;

    for (int y = kernel_size / 2; y < img.rows - kernel_size / 2; y++) {
        for (int x = kernel_size / 2; x < img.cols - kernel_size / 2; x++) {
            int counter = 0;
            for (int k = 0; k < kernel_size; k++) {
                for (int l = 0; l < kernel_size; l++) {
                    window[counter] = img.at<uchar>(y - (kernel_size / 2) + l, x - (kernel_size / 2) + k);
                    counter++;
                }
            }

            if (sort_type == 0) {
                final_img.at<uchar>(y, x) = insertionSort(window, kernel_size);
            }
            else {
                std::sort(window, window);
                final_img.at<uchar>(y, x) = window[kernel_size * kernel_size / 2 + 1]; //Introsort 
            }
        }
    }

    return final_img;
}

int main()
{
    for (size_t i = 512; i < 5000; i=i*2)
    {
        std::string img_size = std::to_string(i);
        int kernel_size = 3;
        cv::Mat img = cv::imread("data/" + img_size + ".png", 0);
        cv::Mat cpu_final_1, cpu_final_2, cpu_final_3, cpu_final_4, cpu_final_5;
        cpu_final_4 = img.clone();
        cpu_final_5 = img.clone();

        auto st1 = std::chrono::high_resolution_clock::now();
        cpu_final_1 = cpu_edition(img, 0, kernel_size);
        auto st2 = std::chrono::high_resolution_clock::now();
        cpu_final_2 = cpu_edition(img, 1, kernel_size);
        auto st3 = std::chrono::high_resolution_clock::now();
        cv::medianBlur(img, cpu_final_3, kernel_size);
        auto st4 = std::chrono::high_resolution_clock::now();
        median_filter_wrapper(img, cpu_final_4, false);
        auto st5 = std::chrono::high_resolution_clock::now();
        median_filter_wrapper(img, cpu_final_5, true);
        auto st6 = std::chrono::high_resolution_clock::now();

        float cpu_insertion_elapsed_time_ms = std::chrono::duration_cast<std::chrono::milliseconds>(st2 - st1).count();
        float cpu_short_elapsed_time_ms = std::chrono::duration_cast<std::chrono::milliseconds>(st3 - st2).count();
        float opencv_time_ms = std::chrono::duration_cast<std::chrono::milliseconds>(st4 - st3).count();
        float gpu_time_ms = std::chrono::duration_cast<std::chrono::milliseconds>(st5 - st4).count();
        float gpu_time_shared_ms = std::chrono::duration_cast<std::chrono::milliseconds>(st6 - st5).count();

        std::cout << "Image size: " << std::to_string(i) << " - Kernel size: " << std::to_string(kernel_size) << std::endl;
        std::cout << "cpu_insertion_ms: " << cpu_insertion_elapsed_time_ms << std::endl;
        std::cout << "cpu_short_ms: " << cpu_short_elapsed_time_ms << std::endl;
        std::cout << "opencv_time_ms: " << opencv_time_ms << std::endl;
        std::cout << "gpu_time_ms: " << gpu_time_ms << std::endl;
        std::cout << "gpu_time_shared_ms: " << gpu_time_shared_ms << std::endl << std::endl;

        imwrite(std::to_string(i) + "_" + std::to_string(kernel_size) + ".png", cpu_final_5);
    }
    


    return 0;
}
